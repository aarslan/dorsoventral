#include "hip/hip_runtime.h"
#raw
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#end raw

static const int   HF        = $HF;    // height filter (when constant)
static const int   WF        = $WF;    // width filter (when constant)
static const int   H         = $H;     // image height 
static const int   W         = $W;     // image  width

///////////////////////////////////////////////////////////////////////////////
#raw
#define CUDA_CHECK_RETURN(value)                                              \
{hipError_t _m_cudaStat = value;if (_m_cudaStat != hipSuccess)              \
{fprintf(stderr,"Error %s at line %d in file %s\n",                           \
hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);                         \
exit(1);}}
#end raw
///////////////////////////////////////////////////////////////////////////////

//-----------------------------------------------------------------------------
__global__ void conv(const float im[H][W], const float filt[HF][WF], float out[H][W]){
   
    int tx = threadIdx.x+ blockIdx.x*blockDim.x;
    int ty = threadIdx.y+ blockIdx.y*blockDim.y;

    float res = 0.0f;
    float im_temp;
    float norm = 0.0f;

    if (tx > WF/2 && tx < W - WF/2 && ty> HF/2 && ty < H - HF/2){
        for (int y=0; y<HF; y++){
            for (int x=0; x<WF; x++){
                im_temp = im[ty+y-HF/2][tx+x-WF/2]; 
                res    += im_temp * filt[y][x];
                //norm   += im_temp*im_temp;
            }
        }
        out[ty][tx] = res;
        //norm = sqrtf(norm);
        //if (norm ==0.0f){ out[ty][tx] = 0.0f;}
        //else{  out[ty][tx] = abs(res/norm);}
    }
}
